#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

__global__ void matrix_elementwise_add_kernel(int nrow, int ncol,
                                              const float *matA,
                                              const float *matB,
                                              float *output) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = row_idx * ncol + col_idx;
  if (row_idx < nrow && col_idx < ncol) {
    output[idx] = matA[idx] + matB[idx];
  }
}

__global__ void matrix_elementwise_add_by_const_kernel(int nrow, int ncol,
                                                       const float *input,
                                                       const float val,
                                                       float *output) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = row_idx * ncol + col_idx;
  if (row_idx < nrow && col_idx < ncol) {
    output[idx] = input[idx] + val;
  }
}

__global__ void matrix_elementwise_multiply_kernel(int nrow, int ncol,
                                                   const float *matA,
                                                   const float *matB,
                                                   float *output) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = row_idx * ncol + col_idx;
  if (row_idx < nrow && col_idx < ncol) {
    output[idx] = matA[idx] * matB[idx];
  }
}

__global__ void matrix_elementwise_multiply_by_const_kernel(int nrow, int ncol,
                                                            const float *input,
                                                            const float val,
                                                            float *output) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = row_idx * ncol + col_idx;
  if (row_idx < nrow && col_idx < ncol) {
    output[idx] = input[idx] * val;
  }
}

__global__ void array_set_kernel(int nrow, int ncol,
                                float *arr,
                                const float val) {
  int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = row_idx * ncol + col_idx;
  if (row_idx < nrow && col_idx < ncol) {
    arr[idx] = val;
  }
}

int DLGpuArraySet(DLArrayHandle arr, float value) { 
  /* TODO: Your code here */
  assert(arr->ndim == 2);
  int nrow = arr->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = arr->shape[1];
  float *input_data = (float *)arr->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  array_set_kernel<<<1, threads, nrow * sizeof(float)>>>(nrow, 
                                                         ncol, 
                                                         input_data, 
                                                         value);
  return 0;
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  return 0;
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  return 0;
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
  /* TODO: Your code here */
  assert(matA->ndim == 2);
  assert(matB->ndim == 2);
  assert(output->ndim == 2);
  assert(matA->shape[0] == matB->shape[0] &&
         matA->shape[1] == matB->shape[1]); 
  int nrow = matA->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = matA->shape[1];
  const float *matA_data = (const float *)matA->data;
  const float *matB_data = (const float *)matB->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_elementwise_add_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, matA_data, matB_data, output_data);
  return 0;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
  /* TODO: Your code here */
  assert(input->ndim == 2);
  assert(output->ndim == 2);
  int nrow = input->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input->shape[1];
  const float *input_data = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_elementwise_add_by_const_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data, val, output_data);
  return 0;
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  /* TODO: Your code here */
  assert(matA->ndim == 2);
  assert(matB->ndim == 2);
  assert(output->ndim == 2);
  assert(matA->shape[0] == matB->shape[0] &&
         matA->shape[1] == matB->shape[1]); 
  int nrow = matA->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = matA->shape[1];
  const float *matA_data = (const float *)matA->data;
  const float *matB_data = (const float *)matB->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_elementwise_multiply_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, matA_data, matB_data, output_data);
  return 0;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
  /* TODO: Your code here */
  assert(input->ndim == 2);
  assert(output->ndim == 2);
  int nrow = input->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input->shape[1];
  const float *input_data = (const float *)input->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_elementwise_multiply_by_const_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data, val, output_data);
  return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  /* TODO: Your code here */
  // Hint: use cublas
  // cublas assume matrix is column major
  return 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  return 0;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
  /* TODO: Your code here */
  return 0;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
  /* TODO: Your code here */
  return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
  assert(input_a->ndim == 2);
  assert(input_b->ndim == 2);
  assert(output->ndim == 1);
  assert(input_a->shape[0] == input_b->shape[0] &&
         input_a->shape[1] == input_b->shape[1]);
  int nrow = input_a->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input_a->shape[1];
  const float *input_data_a = (const float *)input_a->data;
  const float *input_data_b = (const float *)input_b->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_cross_entropy_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, input_data_b, output_data);
  return 0;
}
